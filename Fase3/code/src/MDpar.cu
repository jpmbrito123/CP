/*
 MD.c - a simple molecular dynamics program for simulating real gas properties of Lennard-Jones particles.
 
 Copyright (C) 2016  Jonathan J. Foley IV, Chelsea Sweet, Oyewumi Akinfenwa
 
 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or
 (at your option) any later version.
 
 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.
 
 You should have received a copy of the GNU General Public License
 along with this program.  If not, see <http://www.gnu.org/licenses/>.
 
 Electronic Contact:  foleyj10@wpunj.edu
 Mail Contact:   Prof. Jonathan Foley
 Department of Chemistry, William Paterson University
 300 Pompton Road
 Wayne NJ 07470
 
 */
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include <omp.h>
#include <immintrin.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <iostream>

// Number of particles
int N, NUM_THREADS_PER_BLOCK=500, BLOCKS=10;
double *arrayRGPU, *arrayVGPU, *arrayAGPU, *arrayPotGPU, *matrizesAccGPU, *arrayPSUMGPU;
double *PSUMGPU, *POTGPU, *v2GPU, *kinGPU;

//  Lennard-Jones parameters in natural units!
double sigma = 1.;
double epsilon = 1.;
double PEE = 0.;
double m = 1.;
double kB = 1.;

double NA = 6.022140857e23;
double kBSI = 1.38064852e-23;  // m^2*kg/(s^2*K)


//  Size of box, which will be specified in natural units
double L;

//  Initial Temperature in Natural Units
double Tinit;  //2;
//  Vectors!
//
const int MAXPART=5001;
//  Position
double r[MAXPART][3];
//  Velocity
double v[MAXPART][3];
//  Acceleration
double a[MAXPART][3];
//  Force
double F[MAXPART][3];

// atom type
char atype[10];
//  Function prototypes
//  initialize positions on simple cubic lattice, also calls function to initialize velocities
void initialize();  
//  update positions and velocities using Velocity Verlet algorithm 
//  print particle coordinates to file for rendering via VMD or other animation software
//  return 'instantaneous pressure'
double VelocityVerlet(double dt, int iter, FILE *fp);  
//  Compute Force using F = -dV/dr
//  solve F = ma for use in Velocity Verlet~
//  Numerical Recipes function for generation gaussian distribution
double gaussdist();
//  Initialize velocities according to user-supplied initial Temperature (Tinit)
void initializeVelocities();
//  Compute total potential energy from particle coordinates

//  Compute mean squared velocity from particle velocities
double MeanSquaredVelocity();
//  Compute total kinetic energy from particle mass and velocities
double Kinetic();




void checkCUDAError (const char *msg);

void prepareKernels();
void launchComputeAccelerationsKernels();
double launchVelocityVerletKernels(double dt, int iter, FILE *fp);
double launchMeanSquaredVelocityKernel();
double launchKineticKernel();


int main()
{
    
    //  variable delcarations
    int i;
    double dt, Vol, Temp, Press, Pavg, Tavg, rho;
    double VolFac, TempFac, PressFac, timefac;
    double KE, PE, mvs, gc, Z;
    char trash[10000], prefix[1000], tfn[1000], ofn[1000], afn[1000];
    FILE *infp, *tfp, *ofp, *afp;
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  WELCOME TO WILLY P CHEM MD!\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  ENTER A TITLE FOR YOUR CALCULATION!\n");
    scanf("%s",prefix);
    strcpy(tfn,prefix);
    strcat(tfn,"_traj.xyz");
    strcpy(ofn,prefix);
    strcat(ofn,"_output.txt");
    strcpy(afn,prefix);
    strcat(afn,"_average.txt");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("                  TITLE ENTERED AS '%s'\n",prefix);
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    /*     Table of values for Argon relating natural units to SI units:
     *     These are derived from Lennard-Jones parameters from the article
     *     "Liquid argon: Monte carlo and molecular dynamics calculations"
     *     J.A. Barker , R.A. Fisher & R.O. Watts
     *     Mol. Phys., Vol. 21, 657-673 (1971)
     *
     *     mass:     6.633e-26 kg          = one natural unit of mass for argon, by definition
     *     energy:   1.96183e-21 J      = one natural unit of energy for argon, directly from L-J parameters
     *     length:   3.3605e-10  m         = one natural unit of length for argon, directly from L-J parameters
     *     volume:   3.79499-29 m^3        = one natural unit of volume for argon, by length^3
     *     time:     1.951e-12 s           = one natural unit of time for argon, by length*sqrt(mass/energy)
     ***************************************************************************************/
    
    //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    //  Edit these factors to be computed in terms of basic properties in natural units of
    //  the gas being simulated
    
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("  WHICH NOBLE GAS WOULD YOU LIKE TO SIMULATE? (DEFAULT IS ARGON)\n");
    printf("\n  FOR HELIUM,  TYPE 'He' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR NEON,    TYPE 'Ne' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR ARGON,   TYPE 'Ar' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR KRYPTON, TYPE 'Kr' THEN PRESS 'return' TO CONTINUE\n");
    printf("  FOR XENON,   TYPE 'Xe' THEN PRESS 'return' TO CONTINUE\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    scanf("%s",atype);
    
    if (strcmp(atype,"He")==0) {
        
        VolFac = 1.8399744000000005e-29;
        PressFac = 8152287.336171632;
        TempFac = 10.864459551225972;
        timefac = 1.7572698825166272e-12;
        
    }
    else if (strcmp(atype,"Ne")==0) {
        
        VolFac = 2.0570823999999997e-29;
        PressFac = 27223022.27659913;
        TempFac = 40.560648991243625;
        timefac = 2.1192341945685407e-12;
        
    }
    else if (strcmp(atype,"Ar")==0) {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        //strcpy(atype,"Ar");
        
    }
    else if (strcmp(atype,"Kr")==0) {
        
        VolFac = 4.5882712000000004e-29;
        PressFac = 59935428.40275003;
        TempFac = 199.1817584391428;
        timefac = 8.051563913585078e-13;
        
    }
    else if (strcmp(atype,"Xe")==0) {
        
        VolFac = 5.4872e-29;
        PressFac = 70527773.72794868;
        TempFac = 280.30305642163006;
        timefac = 9.018957925790732e-13;
        
    }
    else {
        
        VolFac = 3.7949992920124995e-29;
        PressFac = 51695201.06691862;
        TempFac = 142.0950000000000;
        timefac = 2.09618e-12;
        strcpy(atype,"Ar");
        
    }
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n                     YOU ARE SIMULATING %s GAS! \n",atype);
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    
    printf("\n  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n  YOU WILL NOW ENTER A FEW SIMULATION PARAMETERS\n");
    printf("  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
    printf("\n\n  ENTER THE INTIAL TEMPERATURE OF YOUR GAS IN KELVIN\n");

    scanf("%lf",&Tinit);
    
    // Make sure temperature is a positive number!
    if (Tinit<0.) {
        printf("\n  !!!!! ABSOLUTE TEMPERATURE MUST BE A POSITIVE NUMBER!  PLEASE TRY AGAIN WITH A POSITIVE TEMPERATURE!!!\n");
        exit(0);
    }
   

    // Convert initial temperature from kelvin to natural units
    Tinit /= TempFac;
 
    
    
    printf("\n\n  ENTER THE NUMBER DENSITY IN moles/m^3\n");
    printf("  FOR REFERENCE, NUMBER DENSITY OF AN IDEAL GAS AT STP IS ABOUT 40 moles/m^3\n");
    printf("  NUMBER DENSITY OF LIQUID ARGON AT 1 ATM AND 87 K IS ABOUT 35000 moles/m^3\n");

    scanf("%lf",&rho);
    
    
    N = 5000;//10*216

   
    Vol = N/(rho*NA);
    

    
    Vol /= VolFac;
    

    //  Limiting N to MAXPART for practical reasons
    if (N>=MAXPART) {
        
        printf("\n\n\n  MAXIMUM NUMBER OF PARTICLES IS %i\n\n  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY \n\n", MAXPART);
        exit(0);
        
    }
    
  
    //  Check to see if the volume makes sense - is it too small?
    //  Remember VDW radius of the particles is 1 natural unit of length
    //  and volume = L*L*L, so if V = N*L*L*L = N, then all the particles
    //  will be initialized with an interparticle separation equal to 2xVDW radius
    if (Vol<N) {
        
        printf("\n\n\n  YOUR DENSITY IS VERY HIGH!\n\n");
        printf("  THE NUMBER OF PARTICLES IS %i AND THE AVAILABLE VOLUME IS %f NATURAL UNITS\n",N,Vol);
        printf("  SIMULATIONS WITH DENSITY GREATER THAN 1 PARTCICLE/(1 Natural Unit of Volume) MAY DIVERGE\n");
        printf("  PLEASE ADJUST YOUR INPUT FILE ACCORDINGLY AND RETRY\n\n");
        exit(0);
    }
    

    // Vol = L*L*L;
    // Length of the box in natural units:
    L = pow(Vol,(1./3));

    //  Files that we can write different quantities to
    tfp = fopen(tfn,"w");     //  The MD trajectory, coordinates of every particle at each timestep
    
    ofp = fopen(ofn,"w");     //  Output of other quantities (T, P, gc, etc) at every timestep
    
    afp = fopen(afn,"w");    //  Average T, P, gc, etc from the simulation
    
    int NumTime;
  
    if (strcmp(atype,"He")==0) {
        
        // dt in natural units of time s.t. in SI it is 5 f.s. for all other gasses
        dt = 0.2e-14/timefac;
        //  We will run the simulation for NumTime timesteps.
        //  The total time will be NumTime*dt in natural units
        //  And NumTime*dt multiplied by the appropriate conversion factor for time in seconds
        NumTime=50000;
    }
    else {
        dt = 0.5e-14/timefac;
        NumTime=200;
        
    }
   
    
    //  Put all the atoms in simple crystal lattice and give them random velocities
    //  that corresponds to the initial temperature we have specified
    initialize();
    prepareKernels();
    
    //  Based on their positions, calculate the ininial intermolecular forces
    //  The accellerations of each particle will be defined from the forces and their
    //  mass, and this will allow us to update their positions via Newton's law
    
    launchComputeAccelerationsKernels();
    
    // Print number of particles to the trajectory file
    fprintf(tfp,"%i\n",N);
    
    //  We want to calculate the average Temperature and Pressure for the simulation
    //  The variables need to be set to zero initially
    Pavg = 0;
    Tavg = 0;
    
    
    int tenp = floor(NumTime/10);
    fprintf(ofp,"  time (s)              T(t) (K)              P(t) (Pa)           Kinetic En. (n.u.)     Potential En. (n.u.) Total En. (n.u.)\n");
    printf("  PERCENTAGE OF CALCULATION COMPLETE:\n  [");
    
    for (i=0; i<NumTime+1; i++) {
        
        //  This just prints updates on progress of the calculation for the users convenience
        if (i==tenp) printf(" 10 |");
        else if (i==2*tenp) printf(" 20 |");
        else if (i==3*tenp) printf(" 30 |");
        else if (i==4*tenp) printf(" 40 |");
        else if (i==5*tenp) printf(" 50 |");
        else if (i==6*tenp) printf(" 60 |");
        else if (i==7*tenp) printf(" 70 |");
        else if (i==8*tenp) printf(" 80 |");
        else if (i==9*tenp) printf(" 90 |");
        else if (i==10*tenp) printf(" 100 ]\n");
        fflush(stdout);
        
        
        // This updates the positions and velocities using Newton's Laws
        // Also computes the Pressure as the sum of momentum changes from wall collisions / timestep
        // which is a Kinetic Theory of gasses concept of Pressure

        //Press = VelocityVerlet(dt, i+1, tfp);
        //Press = launchVelocityVerletKernels(dt, i+1, tfp);

        Press = VelocityVerlet(dt, i+1, tfp);
        Press *= PressFac;
        
        //  !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        //  Now we would like to calculate somethings about the system:
        //  Instantaneous mean velocity squared, Temperature, Pressure
        //  Potential, and Kinetic Energy
        //  We would also like to use the IGL to try to see if we can extract the gas constant

        mvs = MeanSquaredVelocity();
        //mvs = launchMeanSquaredVelocityKernel();

        //mvs = MeanSquaredVelocity();

        KE = Kinetic();
        //KE = launchKineticKernel();
        //KE = Kinetic();
        PE = PEE;
        
        // Temperature from Kinetic Theory
        Temp = m*mvs/(3*kB) * TempFac;
        
        // Instantaneous gas constant and compressibility - not well defined because
        // pressure may be zero in some instances because there will be zero wall collisions,
        // pressure may be very high in some instances because there will be a number of collisions
        gc = NA*Press*(Vol*VolFac)/(N*Temp);
        Z  = Press*(Vol*VolFac)/(N*kBSI*Temp);
        
        Tavg += Temp;
        Pavg += Press;
        
        fprintf(ofp,"  %8.4e  %20.8f  %20.8f %20.8f  %20.8f  %20.8f \n",i*dt*timefac,Temp,Press,KE, PE, KE+PE);
        
        
    }
    
    // Free Allocated GPU Memory
    hipFree(arrayAGPU);
    hipFree(arrayRGPU);
    hipFree(arrayPotGPU);
    hipFree(matrizesAccGPU);
    hipFree(POTGPU);
    //cudaFree(arrayVGPU);
    //cudaFree(arrayPSUMGPU);
    //cudaFree(PSUMGPU);
    //cudaFree(v2GPU);
    //cudaFree(kinGPU);

    // Because we have calculated the instantaneous temperature and pressure,
    // we can take the average over the whole simulation here
    Pavg /= NumTime;
    Tavg /= NumTime;
    Z = Pavg*(Vol*VolFac)/(N*kBSI*Tavg);
    gc = NA*Pavg*(Vol*VolFac)/(N*Tavg);
    fprintf(afp,"  Total Time (s)      T (K)               P (Pa)      PV/nT (J/(mol K))         Z           V (m^3)              N\n");
    fprintf(afp," --------------   -----------        ---------------   --------------   ---------------   ------------   -----------\n");
    fprintf(afp,"  %8.4e  %15.5f       %15.5f     %10.5f       %10.5f        %10.5e         %i\n",i*dt*timefac,Tavg,Pavg,gc,Z,Vol*VolFac,N);
    
    printf("\n  TO ANIMATE YOUR SIMULATION, OPEN THE FILE \n  '%s' WITH VMD AFTER THE SIMULATION COMPLETES\n",tfn);
    printf("\n  TO ANALYZE INSTANTANEOUS DATA ABOUT YOUR MOLECULE, OPEN THE FILE \n  '%s' WITH YOUR FAVORITE TEXT EDITOR OR IMPORT THE DATA INTO EXCEL\n",ofn);
    printf("\n  THE FOLLOWING THERMODYNAMIC AVERAGES WILL BE COMPUTED AND WRITTEN TO THE FILE  \n  '%s':\n",afn);
    printf("\n  AVERAGE TEMPERATURE (K):                 %15.5f\n",Tavg);
    printf("\n  AVERAGE PRESSURE  (Pa):                  %15.5f\n",Pavg);
    printf("\n  PV/nT (J * mol^-1 K^-1):                 %15.5f\n",gc);
    printf("\n  PERCENT ERROR of pV/nT AND GAS CONSTANT: %15.5f\n",100*fabs(gc-8.3144598)/8.3144598);
    printf("\n  THE COMPRESSIBILITY (unitless):          %15.5f \n",Z);
    printf("\n  TOTAL VOLUME (m^3):                      %10.5e \n",Vol*VolFac);
    printf("\n  NUMBER OF PARTICLES (unitless):          %i \n", N);
    
    
    
    
    fclose(tfp);
    fclose(ofp);
    fclose(afp);
    
    return 0;
}


void prepareKernels(){
    hipMalloc(&arrayRGPU, N * 3 * sizeof(double));
    hipMalloc(&arrayAGPU, N * 3 * sizeof(double));
    hipMalloc(&POTGPU, sizeof(double));
    hipMalloc(&arrayPotGPU, (N-1) * sizeof(double));
    hipMalloc(&matrizesAccGPU, (N-1) * N * 3 * sizeof(double));
    //cudaMalloc(&arrayVGPU, N * 3 * sizeof(double));
    //cudaMalloc(&arrayPSUMGPU, N * sizeof(double));
    //cudaMalloc(&PSUMGPU, sizeof(double));
    //cudaMalloc(&v2GPU, sizeof(double));
    //cudaMalloc(&kinGPU, sizeof(double));
    checkCUDAError("Memory Allocation Error!");

    //cudaMemcpy(arrayRGPU, r, N * 3 * sizeof(double), cudaMemcpyHostToDevice);
    //cudaMemcpy(arrayVGPU, v, N * 3 * sizeof(double), cudaMemcpyHostToDevice);
    checkCUDAError("Memory Copy (Host -> Dev) Error!");
}



void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
        printf(msg);
        printf(", ");
        printf(hipGetErrorString(err));
		exit(-1);
	}
}


/* THIS PART IS THE COMPUTEACCELERATIONS KERNEL USING ATOMICADD (FOR SOME REASON THIS SHIT DOESN'T WORK IT GETS STUCK)*/

/*
#if __CUDA_ARCH__ < 600
__device__ double myAtomicAdd(double* address, double val){
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif


__global__
void setAccelerationsKernel(int N, double * arrayAGPU, int NUM_THREADS_PER_BLOCK, int BLOCKS){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = BLOCKS * NUM_THREADS_PER_BLOCK;

    for(; id < N * 3; id += total_threads * 3){
        arrayAGPU[id * 3] = 0;
        arrayAGPU[id * 3 + 1] = 0;
        arrayAGPU[id * 3 + 2] = 0;
    }
}

__global__
void computeAccelerationsKernel(int N, double sigma, double *arrayRGPU, double *arrayAGPU, double *POTGPU, int NUM_THREADS_PER_BLOCK, int BLOCKS) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = BLOCKS * NUM_THREADS_PER_BLOCK;
    int j;
    double f, rSqd, quot,term2;
    double rij[3]; // position of i relative to j
    double rij0_f, rij1_f, rij2_f;

    for (; id < (N - 1) * 3; id += total_threads * 3) {
        double prim= 0., seg= 0., terc=0.;
        for (j = (id + 1) * 3; j < N * 3; j+=3) {
            rSqd = 0;


            //  component-by-componenent position of i relative to j
            rij[0] = arrayRGPU[id * 3] - arrayRGPU[j];
            rij[1] = arrayRGPU[id * 3 + 1] - arrayRGPU[j + 1];
            rij[2] = arrayRGPU[id * 3 + 2] - arrayRGPU[j + 2];
            //  sum of squares of the components
            
            rSqd = rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2];
            quot = rSqd * rSqd * rSqd;
            term2 = sigma / quot;

            //Pot += (term2 * term2 - term2);
            myAtomicAdd(&POTGPU[0], (term2 * term2 - term2));

            f = 24 * (1/(quot*rSqd)) * (2 * (1/quot) -1);

            rij0_f = rij[0] * f;
            rij1_f = rij[1] * f;
            rij2_f = rij[2] * f;
            

            prim += rij0_f;
            seg += rij1_f;
            terc += rij2_f;

            myAtomicAdd(&arrayAGPU[j], -rij0_f);
            myAtomicAdd(&arrayAGPU[j + 1], -rij1_f);
            myAtomicAdd(&arrayAGPU[j + 2], -rij2_f);

            //a[j][0] -= rij0_f;
            //a[j][1] -= rij1_f;
            //a[j][2] -= rij2_f;
            


        }
        myAtomicAdd(&arrayAGPU[id], prim);
        myAtomicAdd(&arrayAGPU[id + 1], seg);
        myAtomicAdd(&arrayAGPU[id + 2], terc);

        //a[i][0] += prim;
        //a[i][1] += seg;
        //a[i][2] += terc;
    }
}



void launchComputeAccelerationsKernels(){
    double eightEpsilon = 8 * epsilon;
    double *Pot;

    //cudaMemcpy(arrayAGPU, a, N * 3 * sizeof(double), cudaMemcpyHostToDevice);
    //checkCUDAError("Error copying a -> arrayAGPU");
    cudaMemcpy(arrayRGPU, r, N * 3 * sizeof(double), cudaMemcpyHostToDevice);
    checkCUDAError("Error copying r -> arrayRGPU");
    cudaMemset(POTGPU, 0, sizeof(double));
    checkCUDAError("Error copying Pot -> POTGPU");


    setAccelerationsKernel<<< BLOCKS, NUM_THREADS_PER_BLOCK >>>(N, arrayAGPU, NUM_THREADS_PER_BLOCK, BLOCKS);
    computeAccelerationsKernel<<< BLOCKS, NUM_THREADS_PER_BLOCK >>>(N, sigma, arrayRGPU, arrayAGPU, POTGPU, NUM_THREADS_PER_BLOCK, BLOCKS);
    checkCUDAError("Error launching kernel computeAccelerations");
    
    cudaMemcpy(Pot, POTGPU, sizeof(double), cudaMemcpyDeviceToHost);
    checkCUDAError("Error copying POTGPU -> Pot");
    cudaMemcpy(a, arrayAGPU, N * 3 * sizeof(double), cudaMemcpyDeviceToHost);
    checkCUDAError("Error copying arrayAGPU -> a");
    PEE = Pot[0] * eightEpsilon;
}*/



__global__
void computeAccelerationsReduceKernel(int N, double *arrayMatrizesAGPU, double *arrayAGPU, int NUM_THREADS_PER_BLOCK, int BLOCKS ) {
    int total_threads = NUM_THREADS_PER_BLOCK * BLOCKS;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id >= N){
        return;
    }

    for(; id < N; id+= total_threads){
        arrayAGPU[id*3] = 0;
        arrayAGPU[id*3 + 1] = 0;
        arrayAGPU[id*3 + 2] = 0;


        for(int i=0; i <= id; i++){
            if(i == N-1) break;
            int index = i * N * 3;

            arrayAGPU[id*3] += arrayMatrizesAGPU[index + (3 * id)];
            arrayAGPU[id*3 + 1] += arrayMatrizesAGPU[index + (3 * id) + 1];
            arrayAGPU[id*3 + 2] += arrayMatrizesAGPU[index + (3 * id) + 2];
        }
    }
}



__global__
void computeAccelerationsMapKernel(int N, double sigma, double *rGPU, double *arrayMatrizesAGPU, double *POTGPU, int NUM_THREADS_PER_BLOCK, int BLOCKS) {
    int i;
    int total_threads = NUM_THREADS_PER_BLOCK * BLOCKS;
    double f, rSqd, quot, term2;
    double rij[3];
    double rij0_f, rij1_f, rij2_f;

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id >= N-1){
        return;
    }
    

    id = (N - 2) - id;
    for(; id >= 0 ; id-= total_threads){
        double prim = 0., seg = 0., terc = 0.;
        int index = id * N * 3;

        POTGPU[id] = 0;


        for(i = 0; i < N * 3 ; i+=3){
            int ind = index + i;
            arrayMatrizesAGPU[ind] = 0;
            arrayMatrizesAGPU[ind + 1] = 0;
            arrayMatrizesAGPU[ind + 2] = 0;
        }

        for (i = (id*3)+3; i < N * 3 ; i+=3){ 
            rSqd = 0;

            rij[0] = rGPU[id * 3] - rGPU[i];
            rij[1] = rGPU[id * 3 + 1] - rGPU[i + 1];
            rij[2] = rGPU[id * 3 + 2] - rGPU[i + 2];

            rSqd = rij[0] * rij[0] + rij[1] * rij[1] + rij[2] * rij[2];
            quot = rSqd * rSqd * rSqd;
            term2 = sigma / quot;


            POTGPU[id] += (term2 * term2 - term2);


            f = 24 * (1 / (quot * rSqd)) * (2 * (1 / quot) - 1);

            rij0_f = rij[0] * f;
            rij1_f = rij[1] * f;
            rij2_f = rij[2] * f;


            prim += rij0_f;
            seg += rij1_f;
            terc += rij2_f;

            arrayMatrizesAGPU[index + i] -= rij0_f;
            arrayMatrizesAGPU[index + i + 1] -= rij1_f;
            arrayMatrizesAGPU[index + i + 2] -= rij2_f;
        }

        arrayMatrizesAGPU[index + (id * 3)] += prim;
        arrayMatrizesAGPU[index + (id * 3) + 1] += seg;
        arrayMatrizesAGPU[index + (id * 3) + 2] += terc;
    }
}




__global__
void calculatePOT(int N, double *arrayPotGPU, double *POTGPU){
    POTGPU[0] = 0.;
    for(int i = 0; i < N - 1; i++) POTGPU[0] += arrayPotGPU[i];
}



void launchComputeAccelerationsKernels(){
    hipMemcpy(arrayRGPU, r, N * 3 * sizeof(double), hipMemcpyHostToDevice);
    checkCUDAError("Error copying arrayAGPU -> a");
    double eightEpsilon = 8 * epsilon;
    double Pot[1];

    computeAccelerationsMapKernel<<< BLOCKS, NUM_THREADS_PER_BLOCK >>>(N, sigma, arrayRGPU, matrizesAccGPU, arrayPotGPU, NUM_THREADS_PER_BLOCK, BLOCKS);
    checkCUDAError("Error launching kernel computeAccelerationsMap");
    computeAccelerationsReduceKernel<<< BLOCKS, NUM_THREADS_PER_BLOCK >>>(N, matrizesAccGPU, arrayAGPU, NUM_THREADS_PER_BLOCK, BLOCKS);
    checkCUDAError("Error launching kernel computeAccelerationsReduce");
    calculatePOT<<<1, 1>>>(N, arrayPotGPU, POTGPU);
    checkCUDAError("Error launching kernel calculatePOT");
    hipMemcpy(Pot, POTGPU, sizeof(double), hipMemcpyDeviceToHost);
    checkCUDAError("Error copying POTGPU -> Pot");
    hipMemcpy(a, arrayAGPU, N* 3* sizeof(double), hipMemcpyDeviceToHost);
    checkCUDAError("Error copying arrayAGPU -> a");
    PEE = Pot[0] * eightEpsilon;
}


// returns sum of dv/dt*m/A (aka Pressure) from elastic collisions with walls
double VelocityVerlet(double dt, int iter, FILE *fp) {
    int i, j;
    
    double psum = 0.;
    
    //  Compute accelerations from forces at current position
    // this call was removed (commented) for predagogical reasons
    //computeAccelerations();
    //  Update positions and velocity with current velocity and acceleration
    //printf("  Updated Positions!\n");
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            r[i][j] += v[i][j]*dt + 0.5*a[i][j]*dt*dt;
            
            v[i][j] += 0.5*a[i][j]*dt;
        }
        //printf("  %i  %6.4e   %6.4e   %6.4e\n",i,r[i][0],r[i][1],r[i][2]);
    }
    //  Update accellerations from updated positions
    launchComputeAccelerationsKernels();
    //  Update velocity with updated acceleration
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            v[i][j] += 0.5*a[i][j]*dt;
        }
    }
    
    // Elastic walls
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            if (r[i][j]<0.) {
                v[i][j] *=-1.; //- elastic walls
                psum += 2*m*fabs(v[i][j])/dt;  // contribution to pressure from "left" walls
            }
            if (r[i][j]>=L) {
                v[i][j]*=-1.;  //- elastic walls
                psum += 2*m*fabs(v[i][j])/dt;  // contribution to pressure from "right" walls
            }
        }
    }
    
    
    /* removed, uncomment to save atoms positions */
    /*for (i=0; i<N; i++) {
        fprintf(fp,"%s",atype);
        for (j=0; j<3; j++) {
            fprintf(fp,"  %12.10e ",r[i][j]);
        }
        fprintf(fp,"\n");
    }*/
    //fprintf(fp,"\n \n");
    
    return psum/(6*L*L);
}



//  Function to calculate the averaged velocity squared
double MeanSquaredVelocity() { 
    
    double vx2 = 0;
    double vy2 = 0;
    double vz2 = 0;
    double v2;

    
    for (int i=0; i<N; i++) {
        
        vx2 = vx2 + v[i][0]*v[i][0];
        vy2 = vy2 + v[i][1]*v[i][1];
        vz2 = vz2 + v[i][2]*v[i][2];
        
    }
    v2 = (vx2+vy2+vz2)/N;
    
    
    //printf("  Average of x-component of velocity squared is %f\n",v2);
    return v2;
}



//  Function to calculate the kinetic energy of the system
double Kinetic() { //Write Function here!  
    
    double v2, kin;

    
    kin =0.;
    for (int i=0; i<N; i++) {
        
        v2 = 0.;
        for (int j=0; j<3; j++) {
            
            v2 += v[i][j]*v[i][j];
            
        }
        //ORIGINAL
        // kin += m*v2/2.;
        //EDITED
        kin += m*v2/2;
        
    }
    
    //printf("  Total Kinetic Energy is %f\n",N*mvs*m/2.);
    return kin;
    
}



void initialize() {
    int n, p, i, j, k;
    double pos;
    
    // Number of atoms in each direction
    n = int(ceil(pow(N, 1.0/3)));
    
    //  spacing between atoms along a given direction
    pos = L / n;
    
    //  index for number of particles assigned positions
    p = 0;
    //  initialize positions
      for (i=0; i<n; i++) {
        for (j=0; j<n; j++) {
            for (k=0; k<n; k++) {
                if (p<N) {
                    
                    r[p][0] = (i + 0.5)*pos;
                    r[p][1] = (j + 0.5)*pos;
                    r[p][2] = (k + 0.5)*pos;
                }
                p++;
            }
        }
    }
   
    // Call function to initialize velocities
    initializeVelocities();
    
    /***********************************************
     *   Uncomment if you want to see what the initial positions and velocities are

     printf("  Printing initial positions!\n");
     for (i=0; i<N; i++) {
     printf("  %6.3e  %6.3e  %6.3e\n",r[i][0],r[i][1],r[i][2]);
     }
     printf("  Printing initial velocities!\n");
     for (i=0; i<N; i++) {
     printf("  %6.3e  %6.3e  %6.3e\n",v[i][0],v[i][1],v[i][2]);
     }
     */
    
    
    
}   



void initializeVelocities() {
    
    int i, j;
    
    for (i=0; i<N; i++) {
        
        for (j=0; j<3; j++) {
            //  Pull a number from a Gaussian Distribution
            v[i][j] = gaussdist();
            
        }
    }
    
    // Vcm = sum_i^N  m*v_i/  sum_i^N  M
    // Compute center-of-mas velocity according to the formula above
    double vCM[3] = {0, 0, 0};
    
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            vCM[j] += m*v[i][j];
            
        }
    }
    
    
    for (i=0; i<3; i++) vCM[i] /= N*m;
    
    //  Subtract out the center-of-mass velocity from the
    //  velocity of each particle... effectively set the
    //  center of mass velocity to zero so that the system does
    //  not drift in space!
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            v[i][j] -= vCM[j];
            
        }
    }
    
    //  Now we want to scale the average velocity of the system
    //  by a factor which is consistent with our initial temperature, Tinit
    double vSqdSum, lambda;
    vSqdSum=0.;
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            vSqdSum += v[i][j]*v[i][j];
            
        }
    }
    
    lambda = sqrt( 3*(N-1)*Tinit/vSqdSum);
    
    for (i=0; i<N; i++) {
        for (j=0; j<3; j++) {
            
            v[i][j] *= lambda;
            
        }
    }
}


//  Numerical recipes Gaussian distribution number generator
double gaussdist() {
    static bool available = false;
    static double gset;
    double fac, rsq, v1, v2;
    if (!available) {
        do {
            v1 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            v2 = 2.0 * rand() / double(RAND_MAX) - 1.0;
            rsq = v1 * v1 + v2 * v2;
        } while (rsq >= 1.0 || rsq == 0.0);
        
        fac = sqrt(-2.0 * log(rsq) / rsq);
        gset = v1 * fac;
        available = true;
        
        return v2*fac;
    } else {
        
        available = false;
        return gset;
        
    }
}




/* THIS KERNELS ARE USED FOR THE VELOCITYVERLET, MEANSQUAREDVELOCITIES AND KINETIC PARTS (COULD HAVE SOME MINOR ERRORS)*/

/*
__global__
void velocityVerletFirstPart(double dt, double *arrayAGPU, double *arrayVGPU, double *arrayRGPU){
    int id = blockIdx.x * blockDim.x + threadIdx.x * 3; 
    
    //  Compute accelerations from forces at current position
    // this call was removed (commented) for predagogical reasons
    //computeAccelerations();
    //  Update positions and velocity with current velocity and acceleration
    //printf("  Updated Positions!\n");
    for (int j=0; j<3; j++) {

        arrayRGPU[id + j] += arrayVGPU[id + j] * dt + 0.5*arrayAGPU[id + j] * dt * dt;
        arrayVGPU[id + j] += 0.5*arrayAGPU[id + j] * dt;
    }
    //printf("  %i  %6.4e   %6.4e   %6.4e\n",i,r[i][0],r[i][1],r[i][2]);
}



__global__
void velocityVerletSecondPart(double L, double dt, double m, double *arrayAGPU, double *arrayVGPU, double *arrayRGPU, double *arrayPSUMGPU){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    arrayPSUMGPU[id] = 0.;

    //  Update velocity with updated acceleration
    for (int j=0; j<3; j++) {
        arrayVGPU[id * 3 + j] += 0.5*arrayAGPU[id * 3 + j]* dt;
    }
    
    // Elastic walls
    for (int j=0; j<3; j++) {

        if (arrayRGPU[id * 3 + j]<0.) {
            arrayVGPU[id * 3 + j] *=-1.; //- elastic walls
            arrayPSUMGPU[id] += 2*m*fabs(arrayVGPU[id * 3 + j]) / dt;  // contribution to pressure from "left" walls
        }

        if (arrayRGPU[id * 3 + j]>=L) {
            arrayVGPU[id * 3 + j]*=-1.;  //- elastic walls
            arrayPSUMGPU[id] += 2*m*fabs(arrayVGPU[id * 3 + j])/dt;  // contribution to pressure from "right" walls
        }
    }
}


__global__
void calculatePSUM(int N, double *arrayPSUMGPU, double *PSUMGPU){
    PSUMGPU[0] = 0.;
    for(int i = 0; i < N; i++) PSUMGPU[0] += arrayPSUMGPU[i];
}



double launchVelocityVerletKernels(double dt, int iter, FILE *fp){
    BLOCKS = 10;
    NUM_THREADS_PER_BLOCK = 500;

    velocityVerletFirstPart<<< BLOCKS, NUM_THREADS_PER_BLOCK >>>(dt, arrayAGPU, arrayVGPU, arrayRGPU);
    checkCUDAError("Error launching kernel velocityVerletFirstPart");
    launchComputeAccelerationsKernels();
    velocityVerletSecondPart<<< BLOCKS, NUM_THREADS_PER_BLOCK >>>(L, dt, m, arrayAGPU, arrayVGPU, arrayRGPU, arrayPSUMGPU);
    checkCUDAError("Error launching kernel velocityVerletSecondPart");
    calculatePSUM<<<1, 1>>>(N, arrayPSUMGPU, PSUMGPU);
    checkCUDAError("Error launching kernel calculatePSUM");

    double psum[1];
    cudaMemcpy(psum, PSUMGPU, sizeof(double), cudaMemcpyDeviceToHost);
    checkCUDAError("Error copying PSUMGPU -> psum");
    cudaMemcpy(v, arrayVGPU, N * 3 * sizeof(double), cudaMemcpyDeviceToHost);
    checkCUDAError("Error copying arrayVGPU -> v");

    return psum[0];
}


__global__
void meanSquaredVelocityKernel(int N, double *arrayVGPU, double *v2GPU){
    double vx2 = 0;
    double vy2 = 0;
    double vz2 = 0;
    
    for (int i=0; i < N * 3; i+=3) {
        
        vx2 = vx2 + arrayVGPU[i]*arrayVGPU[i];
        vy2 = vy2 + arrayVGPU[i + 1]*arrayVGPU[i + 1];
        vz2 = vz2 + arrayVGPU[i + 2]*arrayVGPU[i + 2];
        
    }
    v2GPU[0] = (vx2+vy2+vz2)/N;
}


double launchMeanSquaredVelocityKernel(){
    meanSquaredVelocityKernel<<<1, 1>>>(N, arrayVGPU, v2GPU);
    checkCUDAError("Error launching kernel meanSquaredVelocityKernel");
    double v2[1];
    cudaMemcpy(v2, v2GPU, sizeof(double), cudaMemcpyDeviceToHost);
    checkCUDAError("Error copying v2GPU -> v2");
    return v2[0];
}



__global__
void kineticKernel(int N, double m, double *arrayVGPU, double *kinGPU){
    double v2;

    kinGPU[0] = 0.;
    for (int i=0; i< N * 3; i+=3) {
        
        v2 = 0.;
        for (int j=0; j<3; j++) {
            
            v2 += arrayVGPU[i + j]*arrayVGPU[i + j];
            
        }
        //ORIGINAL
        // kin += m*v2/2.;
        //EDITED
        kinGPU[0] += m*v2/2;
        
    }  
    //printf("  Total Kinetic Energy is %f\n",N*mvs*m/2.);
}


double launchKineticKernel(){
    kineticKernel<<<1, 1>>>(N, m, arrayVGPU, kinGPU);
    checkCUDAError("Error launching kernel kineticKernel");
    double kin[1];
    cudaMemcpy(kin, kinGPU, sizeof(double), cudaMemcpyDeviceToHost);
    checkCUDAError("Error copying kinGPU -> kin");
    return kin[0];
}*/